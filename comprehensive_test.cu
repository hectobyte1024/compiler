#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

// CUDA Kernel: testVertex
__global__ void testVertex_kernel(float* vertices, float* output, int numVertices) {
float3 position    ;
float2 texCoord    ;
float2 vTexCoord    ;
__device__ void main(    ) {
        vTexCoord = texCoord;
        gl_Position = vec4(position, 1.0);
    }
    
}

// CUDA Kernel: testFragment
__global__ void testFragment_kernel(float* pixels, int width, int height) {
float2 vTexCoord    ;
float4 fragColor    ;
__constant__ hipTextureObject_t diffuseTexture    ;
__device__ void main(    ) {
float3 color = texture(diffuseTexture, vTexCoord).rgb        ;
        fragColor = vec4(color, 1.0);
    }
    
}

